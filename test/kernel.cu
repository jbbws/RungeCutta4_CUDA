#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>


hipError_t RungeCutta4(const int size,float step, float a, float b, float *init, float *y1, float *y2, float *y3);

//�������� ����������� ������
__device__ __constant__ float h;
__device__ __constant__ float l;
__device__ __constant__ float r;
//���������� ������������ �����
__device__ int arraySize = 0;
//�������������� ������� ���������� �������� �������  
__device__ float Func(const int number,float x, float y1, float y2, float y3)
{
	switch (number)
	{
	case 0:
		return -(55 + y3)*y1 + 65 * y2;
		break;
	case 1:
		return 0.0785*(y1-y2);
		break;
	case 2:
		return 0.1*y1;
		break;
	default:
		break;
	}
}
//������� ������ � �������
__host__ void showResult(const float  *y1res,const float *y2res,const float *y3res,int size)
{
	float x = 0.0;
	for (int i = 0; i < size; i++)
	{
		printf("Step: %.2f ",x);
		printf(" y1[%d]: %f", i, y1res[i]);
		printf(" y2[%d]: %f", i, y2res[i]);
		printf(" y3[%d]: %f\n", i, y3res[i]);
		x = x + 0.01;
	}
}
//��� ���� ��� �� ��������
__host__ void mallocMemoryOnHost(float *y1, float *y2, float *y3,const int size)
{
	/*y1 = (float*)malloc(size * sizeof(float));
	y2 = (float*)malloc(size * sizeof(float));
	y3 = (float*)malloc(size * sizeof(float));*/
	y1 = new float[size];
	y2 = new float[size];
	y3 = new float[size];
}
__host__ void freeMemoryOnHost(float *y1, float *y2, float *y3)
{
	delete[]y1;
	delete[]y2;
	delete[]y3;
}
//����
__global__ void core(float *y1, float *y2, float *y3)
{
	int id = threadIdx.x;
	float K0, K1, K2, K3;
	for (int i = 1, float x = l; i < arraySize; i++, x = x + h)
	{
		K0 = Func(id, x, y1[i - 1], y2[i - 1], y3[i - 1]);
		K1 = Func(id, x + h / 2, y1[i - 1] + h / 2 * K0, y2[i - 1] + h / 2 * K0, y3[i - 1] + h / 2 * K0);
		K2 = Func(id, x + h / 2, y1[i - 1] + h / 2 * K1, y2[i - 1] + h / 2 * K1, y3[i - 1] + h / 2 * K1);
		K3 = Func(id, x + h, y1[i - 1] + h * K2, y2[i - 1] + h * K2, y3[i - 1] + h * K2);
		__syncthreads();
		switch (id)
		{
		case 0:
			y1[i] = y1[i - 1] + h / 6 * (K0 + 2 * K1 + 2 * K2 + K3);
			break;
		case 1:
			y2[i] = y2[i - 1] + h / 6 * (K0 + 2 * K1 + 2 * K2 + K3);
			break;
		case 2:
			y3[i] = y3[i - 1] + h / 6 * (K0 + 2 * K1 + 2 * K2 + K3);
			break;
		default:
			break;
		}
		__syncthreads();
	}
}
int main()
{
	
	const int countEq = 3;
	const float step = 0.01;
	const float a = 0;
	const float b = 0.1;

	float *y1 = 0, *y2 = 0, *y3 = 0;
	float initValues[countEq] = { 1.0f,1.0f,0.0f };
	int arrSize = static_cast<int>((b - a) / step) + 1;

	//mallocMemoryOnHost(y1, y2, y3, arrSize);
	y1 = new float[arrSize];
	y2 = new float[arrSize];
	y3 = new float[arrSize];

	hipError_t CudaStat = 	RungeCutta4(arrSize,step,a,b,initValues,y1,y2,y3);
	if (CudaStat != hipSuccess)
	{
		fprintf(stderr, "RungeCutta4 failed!");
		return 1;
	}
	CudaStat = hipDeviceReset();
	if (CudaStat != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	showResult(y1, y2, y3, arrSize);

	delete[]y1;
	delete[]y2;
	delete[]y3;
	//freeMemoryOnHost(y1, y2, y3);
	getchar();
	return 0;
}

hipError_t	RungeCutta4(const int size, float step,float a, float b,float *init, float *y1, float *y2, float *y3)
{
	float *dev_y1 = 0;				//������ ��� 1 ���������
	float *dev_y2 = 0;				//������ ��� 2 ���������
	float *dev_y3 = 0;				//������ ��� 3 ���������
	hipError_t	status;
	//printf("Size is: %d", size);
	status = hipSetDevice(0);	//�������� ����������
	if (status != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	status = hipMalloc((void**)&dev_y1, size * sizeof(float));			//�������� ������ ��� 1-��� �������
	status = hipMalloc((void**)&dev_y2, size * sizeof(float));			//�������� ������ ��� 2-��� �������
	status = hipMalloc((void**)&dev_y3, size * sizeof(float));			//�������� ������ ��� 3-��� �������
	if (status != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	status = hipMemcpyToSymbol(HIP_SYMBOL(h), &step, sizeof(float), 0, hipMemcpyHostToDevice);//�������� � �����.������ �������� ����
	status = hipMemcpyToSymbol(HIP_SYMBOL(l), &a, sizeof(float), 0, hipMemcpyHostToDevice);//�������� � �����.������ �������� ����� �������
	status = hipMemcpyToSymbol(HIP_SYMBOL(r), &b, sizeof(float), 0, hipMemcpyHostToDevice);//�������� � �����.������ �������� ������ �������
	status = hipMemcpyToSymbol(HIP_SYMBOL(arraySize), &size, sizeof(int), 0, hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpyToSymbol failed!");
		goto Error;
	}
	//�������� � ������ ��������� �������� ��� ������ ����
	status = hipMemcpy(dev_y1, &init[0], sizeof(float), hipMemcpyHostToDevice); 
	status = hipMemcpy(dev_y2, &init[1], sizeof(float), hipMemcpyHostToDevice);
	status = hipMemcpy(dev_y3, &init[2], sizeof(float), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy init values failed!");
		goto Error;
	}

	core<<<1, 3 >>> (dev_y1, dev_y2, dev_y3);
	status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "core launch failed: %s\n", hipGetErrorString(status));
		goto Error;
	}
	status = hipDeviceSynchronize();
	if (status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", status);
		goto Error;
	}
	status = hipMemcpy(y1, dev_y1, size * sizeof(float), hipMemcpyDeviceToHost);
	status = hipMemcpy(y2, dev_y2, size * sizeof(float), hipMemcpyDeviceToHost);
	status = hipMemcpy(y3, dev_y3, size * sizeof(float), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpyDeviceToHost failed!");
		goto Error;
	}
Error:
	hipFree(dev_y1);
	hipFree(dev_y2);
	hipFree(dev_y3);
	return status;
}
